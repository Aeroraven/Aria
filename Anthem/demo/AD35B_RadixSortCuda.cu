#include "hip/hip_runtime.h"
#include "../include/core/base/AnthemDefs.h"
#include "../include/core/base/AnthemLogger.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

typedef uint32_t u32;
typedef uint64_t u64;
typedef int32_t i32;
typedef int64_t i64;


#define DCONST const
DCONST u32 LOG_BLOCK_SIZE = 10;
DCONST u32 BLOCK_SIZE = (1 << LOG_BLOCK_SIZE);
DCONST u32 LOG_NUM_BLOCKS = 8;
DCONST u32 NUM_BLOCKS = 1 << LOG_NUM_BLOCKS;
DCONST u32 TOTAL_SIZE = BLOCK_SIZE * NUM_BLOCKS;
DCONST u32 BIT_PER_PASS = 2;
DCONST u32 BLOCK_SUM_SIZE = NUM_BLOCKS * (1 << BIT_PER_PASS);
DCONST u32 MAX_BITS = 32;
DCONST u32 BIT_CANDIDATES = 1<<BIT_PER_PASS;
DCONST char* SAVE_RESULT = ANTH_ASSET_TEMP_DIR"radixresultcuda.txt";
#undef DCONST


u32* hData;
u32* hResult;
u32* hDataCopy;
u32* dBufferGlobal[2];
u32* dLocalPrefixSum;
u32* dBlockPrefixSum;

double cudaRadixTime = 0.0;
double cudaRadixTimeAll = 0.0;
double stdSortTime = 0.0;
double cpuRadixTime = 0.0;

struct CPURadix {
	// Code enclosed in this struct from: 
	// https://www.geeksforgeeks.org/radix-sort/
	int getMax(u32 a[], int n) {
		u32 i, max = 0;
		for (i = 0; i < n; i++) {
			if (max < a[i])max = a[i];
		}
		return max;
	}
	void countSort(u32 a[], int n, int exp) {
		u32* output = new u32[n];
		u32 i, buckets[10] = { 0 };
		u32* temp = new u32[n];
		for (int i = 0; i < n; i++) {
			buckets[(a[i] / exp) % 10]++;
		}
		for (int i = 1; i < 10; i++) {
			buckets[i] += buckets[i - 1];
		}
		for (int i = n - 1; i >= 0; i--) {
			output[buckets[(a[i] / exp) % 10] - 1] = a[i];
			buckets[(a[i] / exp) % 10]--;
		}
		for (int i = 0; i < n; i++) {
			a[i] = output[i];
		}
	}
	void radixSort(u32 a[], int n) {
		u32 exp;
		u32 max = getMax(a, n);
		for (exp = 1; max / exp > 0; exp *= 10) {
			countSort(a, n, exp);
		}
	}
};

__global__ void radixSortLocalHistogram(const int iter,u32* dLocalPrefixSum,u32* dBlockPrefixSum,u32* dSrc) {
	u32 tId = threadIdx.x;
	u32 bId = blockIdx.x;
	u32 gId = bId * BLOCK_SIZE + tId;
	u32* dBuffer = dSrc;
	__shared__ u32 localBins[BLOCK_SIZE];
	__shared__ u32 localCounter[BIT_CANDIDATES];

	if(tId<BIT_CANDIDATES){
		localCounter[tId] = 0;
	}
	__syncthreads();

	u32 curVal = dBuffer[gId];
	u32 curBit = (curVal>>(iter*BIT_PER_PASS)) & (BIT_CANDIDATES-1);

	atomicAdd(&localCounter[curBit], 1);
	__syncthreads();

	for(i32 i=0;i<BIT_CANDIDATES;i++){
		localBins[tId] = (curBit==i)?1:0;
		
		__syncthreads();
		for(i32 j=0;j<LOG_BLOCK_SIZE;j++){
			u32 base = (1<<(j+1))-1;
			u32 increment = (1<<(j+1));
			if(tId>=base&&(tId-base)%increment==0){
				localBins[tId] += localBins[tId-increment/2];
			}
			__syncthreads();
		}
		for(i32 j= LOG_BLOCK_SIZE-2;j>=0;j--){
			u32 base = (1<<(j+1))-1+(1<<(j));
			u32 increment = (1<<(j+1));
			if(tId>=base&&(tId-base)%increment==0){
				localBins[tId] += localBins[tId-increment/2];
			}
			__syncthreads();
		}
		if(curBit==i){
			dLocalPrefixSum[gId] = localBins[tId]-1;
		}
		__syncthreads();
	}
	if(tId<BIT_CANDIDATES){
		dBlockPrefixSum[tId*NUM_BLOCKS+bId] = localCounter[tId];
	}
}

__global__ void radixSortBlockPrefix(u32* dBlockPrefixSum){
	u32 tId = threadIdx.x;
	__shared__ u32 localCounter[NUM_BLOCKS * BIT_CANDIDATES];
	__shared__ u32 srcValue[NUM_BLOCKS * BIT_CANDIDATES];

	srcValue[tId] = dBlockPrefixSum[tId];
	localCounter[tId] = srcValue[tId];
	__syncthreads();
	for(i32 i=0;i<BIT_PER_PASS*LOG_NUM_BLOCKS;i++){
		u32 base = (1<<(i+1))-1;
		u32 increment = (1<<(i+1));
		if(tId>=base&&(tId-base)%increment==0){
			localCounter[tId] += localCounter[tId-increment/2];
		}
		__syncthreads();
	}
	for(i32 i= BIT_PER_PASS * LOG_NUM_BLOCKS -2;i>=0;i--){
		u32 base = (1<<(i+1))-1+(1<<(i));
		u32 increment = (1<<(i+1));
		if(tId>=base&&(tId-base)%increment==0){
			localCounter[tId] += localCounter[tId-increment/2];
		}
		__syncthreads();
	}
	dBlockPrefixSum[tId] = localCounter[tId] - srcValue[tId];
}

__global__ void radixSortScatter(const int iter,u32* dBlockPrefixSum,u32* dLocalPrefixSum, u32* dSrc,u32* dDst) {
	u32 tId = threadIdx.x;
	u32 bId = blockIdx.x;
	u32 gId = bId * BLOCK_SIZE + tId;
	u32* dBuffer = dSrc;

	u32 curVal = dBuffer[gId];
	u32 curBit = (curVal>>(iter*BIT_PER_PASS)) & (BIT_CANDIDATES-1);
	u32 blockPrefix = dBlockPrefixSum[curBit* NUM_BLOCKS +bId];
	u32 localPrefix = dLocalPrefixSum[gId];
	u32 scatterPos = blockPrefix + localPrefix;
	dDst[scatterPos] = curVal;
}

void radixSortExec(){
	for(i32 i=0;i<MAX_BITS/BIT_PER_PASS;i++){
		radixSortLocalHistogram AT_KARG2(NUM_BLOCKS, BLOCK_SIZE) (i,dLocalPrefixSum,dBlockPrefixSum,dBufferGlobal[i & 1]);
		radixSortBlockPrefix AT_KARG2(1, NUM_BLOCKS * BIT_CANDIDATES) (dBlockPrefixSum);
		radixSortScatter AT_KARG2(NUM_BLOCKS, BLOCK_SIZE) (i,dBlockPrefixSum,dLocalPrefixSum, dBufferGlobal[i & 1], dBufferGlobal[1-(i & 1)]);
	}
}

void radixSort() {

	hipMalloc(&dBufferGlobal[0], TOTAL_SIZE * sizeof(u32));
	hipMalloc(&dBufferGlobal[1], TOTAL_SIZE * sizeof(u32));
	hipMalloc(&dLocalPrefixSum, TOTAL_SIZE * sizeof(u32));
	hipMalloc(&dBlockPrefixSum, BLOCK_SUM_SIZE * sizeof(u32));
	auto start1 = std::chrono::high_resolution_clock::now();
	hipMemcpy(dBufferGlobal[0], hData, TOTAL_SIZE * sizeof(u32), hipMemcpyHostToDevice);
	auto start = std::chrono::high_resolution_clock::now();
	radixSortExec();
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();
	cudaRadixTime = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
	//hipMemcpy(hResult,dBlockPrefixSum, NUM_BLOCKS * BIT_CANDIDATES * sizeof(u32), hipMemcpyDeviceToHost);
	hipMemcpy(hResult, dBufferGlobal[MAX_BITS/BIT_PER_PASS & 1], TOTAL_SIZE*sizeof(u32), hipMemcpyDeviceToHost);
	auto end1 = std::chrono::high_resolution_clock::now();
	cudaRadixTimeAll = std::chrono::duration_cast<std::chrono::microseconds>(end1 - start1).count();

	hipFree(dBufferGlobal[0]);
	hipFree(dBufferGlobal[1]);
	hipFree(dLocalPrefixSum);
	hipFree(dBlockPrefixSum);

}

void testRadixSort() {
	hData = new u32[TOTAL_SIZE];
	hResult = new u32[TOTAL_SIZE];
	for (i32 i = 0; i < TOTAL_SIZE; i++) {
		hData[i] = (rand()%32767)*(rand()%65536);
	}

	// CUDA Radix Sort
	radixSort();
	FILE* fp = fopen(SAVE_RESULT, "w");
	for (i32 i = 0; i < TOTAL_SIZE; i++) {
		fprintf(fp, "%d ", hResult[i]);
		if(i%64==63){
			fprintf(fp, "\n");
		}
	}

	// Std::sort
	hDataCopy = new u32[TOTAL_SIZE];
	memcpy(hDataCopy, hData, TOTAL_SIZE * sizeof(u32));
	auto start = std::chrono::high_resolution_clock::now();
	std::sort(hDataCopy, hDataCopy + TOTAL_SIZE);
	auto end = std::chrono::high_resolution_clock::now();
	stdSortTime = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

	// CPU Radix Sort
	CPURadix cpuRadix;
	memcpy(hDataCopy, hData, TOTAL_SIZE * sizeof(u32));
	start = std::chrono::high_resolution_clock::now();
	cpuRadix.radixSort(hDataCopy, TOTAL_SIZE);
	end = std::chrono::high_resolution_clock::now();
	cpuRadixTime = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();


	// Result
	ANTH_LOGI("CUDA Radix Sort Time (Kernel):", cudaRadixTime / 1000.0, " ms");
	ANTH_LOGI("CUDA Radix Sort Time (Total) :", cudaRadixTimeAll / 1000.0, " ms");
	ANTH_LOGI("Std::sort Time               :", stdSortTime / 1000.0, " ms");
	ANTH_LOGI("CPU Radix Sort Time          :", cpuRadixTime / 1000.0, " ms");

	delete[] hData;
	delete[] hResult;
	delete[] hDataCopy;

}

int main() {
	for (i32 i = 0; i < 5; i++) {
		ANTH_LOGI("Test ",i);
		testRadixSort();
	}
	hipDeviceReset();
	return 0;
}