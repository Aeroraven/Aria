#include "hip/hip_runtime.h"
#include "../include/core/base/AnthemDefs.h"
#include <cstdio>
#include "hip/hip_runtime.h"
#include ""


__global__ void helloWorld() {
	printf("%s\n", "Hello World,CUDA");
}

int main() {
	helloWorld AT_KARG2(2,2) ();
	printf("%s\n", "Hello World,x86");
	hipDeviceReset();
	return 0;
}